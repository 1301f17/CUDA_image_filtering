#include "hip/hip_runtime.h"
/* ------------
 * This code is provided solely for the personal and private use of 
 * students taking the CSC367H1 course at the University of Toronto.
 * Copying for purposes other than this use is expressly prohibited. 
 * All forms of distribution of this code, whether as given or with 
 * any changes, are expressly prohibited. 
 * 
 * Authors: Bogdan Simion, Felipe de Azevedo Piovezan
 * 
 * All of the files in this directory and all subdirectories are:
 * Copyright (c) 2017 Bogdan Simion
 * -------------
*/

#include "kernels.h"

__global__ void kernel4(const int8_t *filter, int32_t dimension, 
        const int32_t *input, int32_t *output, int32_t width, int32_t height)
{
    // Number of threads in a grid
    int stride = gridDim.x * blockDim.x;
    int begin_idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (int idx = begin_idx; idx < width * height; idx += stride) {
        int row = idx / width;
        int column = idx % width;
        output[idx] = apply2d_gpu(filter, dimension, input, output, width, height, row, column);
    }
}

__global__ void normalize4(int32_t *image, int32_t width, int32_t height,
        int32_t smallest, int32_t biggest)
{
    // Number of threads in a grid
    int stride = gridDim.x * blockDim.x;
    int begin_idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (int idx = begin_idx; idx < width * height && biggest != smallest; idx += stride) {
        image[idx] = ((image[idx] - smallest) * 255) / (biggest - smallest);
    }
}
