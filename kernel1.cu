#include "hip/hip_runtime.h"
/* ------------
 * This code is provided solely for the personal and private use of 
 * students taking the CSC367H1 course at the University of Toronto.
 * Copying for purposes other than this use is expressly prohibited. 
 * All forms of distribution of this code, whether as given or with 
 * any changes, are expressly prohibited. 
 * 
 * Authors: Bogdan Simion, Felipe de Azevedo Piovezan
 * 
 * All of the files in this directory and all subdirectories are:
 * Copyright (c) 2017 Bogdan Simion
 * -------------
*/

#include "kernels.h"


__global__ void kernel1(const int8_t *filter, int32_t dimension, 
        const int32_t *input, int32_t *output, int32_t width, int32_t height)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int row = index % height;
	int column = index / height;
	int image_index = row * width + column;
	if (index < width * height) {
		output[image_index] = apply2d_gpu(filter, dimension, input, output, width, height, row, column);
	}
}

__global__ void normalize1(int32_t *image, int32_t width, int32_t height,
        int32_t smallest, int32_t biggest)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int row = index % height;
	int column = index / height;
	int image_index = row * width + column;
	if (index < width * height && biggest != smallest) {
		image[image_index] = ((image[image_index] - smallest) * 255) / (biggest - smallest);
	}	
}
